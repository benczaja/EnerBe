#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "../helper.h" // local helper header to clean up code
#include "kernals.h"


int main( int argc, char *argv[] )  {
    
    clock_t t; // declare clock_t (long type)
    kernal kernal;
    kernal.name = "axpy";

    /* VERY DUMB Argument Parsers */
    kernal.size = parse_arguments(argc, argv);

    X_TYPE *d_sx; /* n is an array of N integers */
    X_TYPE *d_sy; /* n is an array of N integers */

    X_TYPE a = 2.0;
    // Allocate Host memory 
    X_TYPE* sx = (X_TYPE*)malloc(kernal.size * sizeof(X_TYPE));
    X_TYPE* sy = (X_TYPE*)malloc(kernal.size * sizeof(X_TYPE));


    // Allocate device memory 
    hipMalloc((void**)&d_sx, sizeof(X_TYPE) * kernal.size);
    hipMalloc((void**)&d_sy, sizeof(X_TYPE) * kernal.size);
        
        
    kernal.algorithm = "simple_gpu";
    hipGetDevice(&kernal.gpuid);  
    int block_size = 512;
    int grid_size = ((kernal.size + block_size) / block_size);

if (true == simple)
  {
    do{
    kernal.start = double(clock()); // start the clock

    // Transfer data from host to device memory
    hipMemcpy(d_sx, sx, sizeof(X_TYPE) * kernal.size, hipMemcpyHostToDevice);
    hipMemcpy(d_sy, sy, sizeof(X_TYPE) * kernal.size, hipMemcpyHostToDevice);

    gpu_axpy<<<grid_size,block_size>>>(kernal.size, a, d_sx, d_sy);

    hipMemcpy(sy, d_sy, sizeof(X_TYPE) * kernal.size, hipMemcpyDeviceToHost);
    
    kernal.end = double(clock()); // stop the clock
      kernal.times[kernal.N_runs] =  (kernal.end - kernal.start)/CLOCKS_PER_SEC;
      kernal.N_runs ++;
    }while (kernal.time < kernal.max_time && kernal.N_runs < kernal.max_runs);
    kernal.calculate_stats();
  }
  kernal.print_info();

    hipFree(d_sx);
    hipFree(d_sy);

    free(sx);
    free(sy);
}
