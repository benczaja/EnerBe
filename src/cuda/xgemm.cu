#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "../helper.h" // local helper header to clean up code
#include "kernals.h"

int main( int argc, char *argv[] )  {

  kernal kernal;
  kernal.name = "xgemm";
  clock_t t; // declare clock_t (long type)

  /* VERY DUMB Argument Parsers */
  kernal.size = parse_arguments(argc, argv, &simple, &openmp, &sanity_check);

  /* declare the arrays...  better to do it as 1D arrays for CUDA */
  // First allocated them on the host (CPU)
  X_TYPE* A = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));
  X_TYPE* B = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));
  X_TYPE* C = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
  X_TYPE* D_A;
  X_TYPE* D_B;
  X_TYPE* D_C;
  hipMalloc((void**)&D_A, sizeof( X_TYPE ) * (kernal.size * kernal.size));
  hipMalloc((void**)&D_B, sizeof( X_TYPE ) * (kernal.size * kernal.size));
  hipMalloc((void**)&D_C, sizeof( X_TYPE ) * (kernal.size * kernal.size));

  double start = omp_get_wtime();  

  initialize_matrix_1D(A, B, C, kernal.size, kernal.size);
    
  double end = omp_get_wtime(); 
  printf("Init TIME: %f s\n",(end-start));


  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/

  /* Simple matrix multiplication */
  /*==============================*/
    kernal.algorithm = "simple_gpu";
    hipGetDevice(&kernal.gpuid);  	

    int block_size = 512;
    int grid_size = ((kernal.size + block_size) / block_size);
    
    t = clock(); // start the clock

    // Transfer data from host to device memory
    hipMemcpy(D_A, A, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyHostToDevice);
    
    simple_matrix_multiply<<<grid_size,block_size>>>(D_A, D_B, D_C, kernal.size, kernal.size);

   // Transfer data from device to host memory
    hipMemcpy(C, D_C, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyDeviceToHost);

    t = clock() - t; // stop the clock

    kernal.time = ((double)t)/CLOCKS_PER_SEC; // convert to seconds (and long to double)

    kernal.print_info();
  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

 // Deallocate device memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

  // Deallocate host memory
  free(A);
  free(B);
  free(C);
}
