#include "hip/hip_runtime.h"
#include <stdio.h> // needed for ‘printf’ 
#include <stdlib.h> // needed for ‘RAND_MAX’ 
#include <omp.h> // needed for OpenMP 
#include <time.h> // needed for clock() and CLOCKS_PER_SEC etc
#include "../helper.h" // local helper header to clean up code
#include <pmt.h> // needed for PMT
#include <iostream> // needed for CPP IO ... cout, endl etc etc
#include "kernals.h"


int main( int argc, char *argv[] )  {

  kernal kernal;
  kernal.name = "xgemm";

  /* VERY DUMB Argument Parsers */
  kernal.size = parse_arguments(argc, argv);
  /* declare the arrays...  better to do it as 1D arrays for CUDA */

  // First allocated them on the host (CPU)
    X_TYPE* A = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));
    X_TYPE* B = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));
    X_TYPE* C = (X_TYPE*)malloc((kernal.size * kernal.size) * sizeof(X_TYPE));

  // Then Allocate them on the GPUs
  X_TYPE* D_A;
  X_TYPE* D_B;
  X_TYPE* D_C;
  hipMalloc((void**)&D_A, sizeof( X_TYPE ) * (kernal.size * kernal.size));
  hipMalloc((void**)&D_B, sizeof( X_TYPE ) * (kernal.size * kernal.size));
  hipMalloc((void**)&D_C, sizeof( X_TYPE ) * (kernal.size * kernal.size));

  double start = omp_get_wtime();  

  initialize_matrix_1D(A, B, C, kernal.size, kernal.size);
    
  double end = omp_get_wtime(); 
  printf("Init TIME: %f sec\n",(end-start));

  // THIS IS NEW !!!!!!!
  std::unique_ptr<pmt::PMT> GPUsensor = pmt::nvml::NVML::Create();
  std::unique_ptr<pmt::PMT> CPUsensor = pmt::rapl::Rapl::Create();
  //Start the PMT "sensor"
  auto GPUstart = GPUsensor->Read(); 
  auto CPUstart = CPUsensor->Read(); 
  auto GPUend = GPUsensor->Read(); 
  auto CPUend = CPUsensor->Read(); 

  /*======================================================================*/
  /*                START of Section of the code that matters!!!          */
  /*======================================================================*/

  /* Simple matrix multiplication */
  /*==============================*/
    if (true ==simple){
    kernal.algorithm = "simple_gpu";
    hipGetDevice(&kernal.gpuid);  	
  
    int block_size = 512;
    int grid_size = ((kernal.size + block_size) / block_size);
    
    do {
    //Start the PMT "sensor"
    GPUstart = GPUsensor->Read(); // READING the GPU via NVML
    CPUstart = CPUsensor->Read(); // READING the CPU via RAPL

    // Transfer data from host to device memory
    hipMemcpy(D_A, A, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyHostToDevice);
    hipMemcpy(D_B, B, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyHostToDevice);
    
    simple_matrix_multiply<<<grid_size,block_size>>>(D_A, D_B, D_C, kernal.size, kernal.size);

   // Transfer data from device to host memory
    hipMemcpy(C, D_C, sizeof(X_TYPE) * (kernal.size * kernal.size), hipMemcpyDeviceToHost);

    //END the PMT "sensor"
    GPUend = GPUsensor->Read();
    CPUend = CPUsensor->Read();

    kernal.rapl_times[kernal.N_runs] = pmt::PMT::seconds(CPUstart, CPUend);
    kernal.rapl_powers[kernal.N_runs] = pmt::PMT::watts(CPUstart, CPUend);
    kernal.rapl_energys[kernal.N_runs] = pmt::PMT::joules(CPUstart, CPUend);

    kernal.nvml_times[kernal.N_runs] = pmt::PMT::seconds(GPUstart, GPUend);
    kernal.nvml_powers[kernal.N_runs] = pmt::PMT::watts(GPUstart, GPUend);
    kernal.nvml_energys[kernal.N_runs] = pmt::PMT::joules(GPUstart, GPUend);

    kernal.N_runs ++;
    }while (kernal.time < kernal.max_time && kernal.N_runs < kernal.max_runs);
    kernal.calculate_stats();
  }
    kernal.print_pmt_nvml_info();


  /*======================================================================*/
  /*                 END of Section of the code that matters!!!           */
  /*======================================================================*/

 // Deallocate device memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);

  // Deallocate host memory
  free(A);
  free(B);
  free(C);
}
